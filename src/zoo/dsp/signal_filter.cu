#include "hip/hip_runtime.h"
#include <include/zoo/dsp/signal_filter.h>
#include <stdexcept>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CHECK_CUDA(call) { const hipError_t e = call; if (e != hipSuccess) { throw std::runtime_error("CUDA Error: " + std::string(hipGetErrorString(e))); } }
#define CHECK_CUFFT(call) { const hipfftResult_t s = call; if (s != HIPFFT_SUCCESS) { throw std::runtime_error("cuFFT Error"); } }

namespace xinfer::zoo::dsp {

__global__ void complex_multiply_kernel(hipfftComplex* a, const hipfftComplex* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float real_a = a[i].x;
        float imag_a = a[i].y;
        float real_b = b[i].x;
        float imag_b = b[i].y;
        a[i].x = real_a * real_b - imag_a * imag_b;
        a[i].y = real_a * imag_b + imag_a * real_b;
    }
}

__global__ void normalize_kernel(float* signal, float norm_factor, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        signal[i] *= norm_factor;
    }
}

struct SignalFilter::Impl {
    SignalFilterConfig config_;
    hipfftHandle fft_plan_;
    int fft_size_;
    float* d_kernel_ = nullptr;
    hipfftComplex* d_kernel_fft_ = nullptr;

    Impl(const SignalFilterConfig& config) : config_(config) {
        if (config.filter_length % 2 == 0) {
            throw std::invalid_argument("Filter length must be odd.");
        }
    }

    ~Impl() {
        hipFree(d_kernel_);
        hipFree(d_kernel_fft_);
        if (fft_plan_) {
            hipfftDestroy(fft_plan_);
        }
    }
};

SignalFilter::SignalFilter(const SignalFilterConfig& config) : pimpl_(new Impl(config)) {}
SignalFilter::~SignalFilter() = default;
SignalFilter::SignalFilter(SignalFilter&&) noexcept = default;
SignalFilter& SignalFilter::operator=(SignalFilter&&) noexcept = default;

std::vector<float> SignalFilter::process(const std::vector<float>& input_signal) {
    if (!pimpl_) throw std::runtime_error("SignalFilter is in a moved-from state.");

    const int signal_len = input_signal.size();
    const int kernel_len = pimpl_->config_.filter_length;
    pimpl_->fft_size_ = 1;
    while (pimpl_->fft_size_ < (signal_len + kernel_len - 1)) {
        pimpl_->fft_size_ *= 2;
    }

    CHECK_CUFFT(hipfftPlan1d(&pimpl_->fft_plan_, pimpl_->fft_size_, HIPFFT_R2C, 1));

    std::vector<float> h_kernel(kernel_len, 0.0f);
    float fc1 = pimpl_->config_.cutoff_freq1 / pimpl_->config_.sample_rate;
    float fc2 = pimpl_->config_.cutoff_freq2 / pimpl_->config_.sample_rate;
    int center = kernel_len / 2;
    for (int i = 0; i < kernel_len; ++i) {
        int n = i - center;
        if (n == 0) {
            h_kernel[i] = 2.0f * M_PI * fc1;
        } else {
            h_kernel[i] = sinf(2.0f * M_PI * fc1 * n) / n;
        }
        h_kernel[i] *= (0.54f - 0.46f * cosf(2.0f * M_PI * i / (kernel_len - 1)));
    }

    if (pimpl_->config_.type == FilterType::HIGH_PASS) {
        for (int i = 0; i < kernel_len; ++i) h_kernel[i] = -h_kernel[i];
        h_kernel[center] += 1.0f;
    }

    CHECK_CUDA(hipMalloc(&pimpl_->d_kernel_, pimpl_->fft_size_ * sizeof(float)));
    CHECK_CUDA(hipMemset(pimpl_->d_kernel_, 0, pimpl_->fft_size_ * sizeof(float)));
    CHECK_CUDA(hipMemcpy(pimpl_->d_kernel_, h_kernel.data(), kernel_len * sizeof(float), hipMemcpyHostToDevice));

    const int fft_complex_size = (pimpl_->fft_size_ / 2) + 1;
    CHECK_CUDA(hipMalloc(&pimpl_->d_kernel_fft_, fft_complex_size * sizeof(hipfftComplex)));
    CHECK_CUFFT(hipfftExecR2C(pimpl_->fft_plan_, pimpl_->d_kernel_, pimpl_->d_kernel_fft_));

    float* d_signal;
    CHECK_CUDA(hipMalloc(&d_signal, pimpl_->fft_size_ * sizeof(float)));
    CHECK_CUDA(hipMemset(d_signal, 0, pimpl_->fft_size_ * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_signal, input_signal.data(), signal_len * sizeof(float), hipMemcpyHostToDevice));

    hipfftComplex* d_signal_fft;
    CHECK_CUDA(hipMalloc(&d_signal_fft, fft_complex_size * sizeof(hipfftComplex)));
    CHECK_CUFFT(hipfftExecR2C(pimpl_->fft_plan_, d_signal, d_signal_fft));

    int threads = 256;
    int blocks = (fft_complex_size + threads - 1) / threads;
    complex_multiply_kernel<<<blocks, threads>>>(d_signal_fft, pimpl_->d_kernel_fft_, fft_complex_size);

    hipfftHandle ifft_plan;
    CHECK_CUFFT(hipfftPlan1d(&ifft_plan, pimpl_->fft_size_, HIPFFT_C2R, 1));
    CHECK_CUFFT(hipfftExecC2R(ifft_plan, d_signal_fft, d_signal));

    normalize_kernel<<< (pimpl_->fft_size_ + threads - 1) / threads, threads >>>(d_signal, 1.0f / pimpl_->fft_size_, pimpl_->fft_size_);

    std::vector<float> filtered_signal(signal_len);
    CHECK_CUDA(hipMemcpy(filtered_signal.data(), d_signal + center, signal_len * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_signal);
    hipFree(d_signal_fft);
    hipfftDestroy(ifft_plan);

    return filtered_signal;
}

} // namespace xinfer::zoo::dsp